#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <algorithm>
#include <string>
#include <unistd.h>
#include <iomanip>

#include "galeshapley.h"

__global__ void stable_matching(int n, int *men_pref, int *women_pref, int *male_match, int *woman_match, 
                                int *propose_next, int *is_stable, int *is_stable_global, int *women_lock,
                                int *d_is_stable_per_block) {

    int m_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (m_idx >= n) {
    //     return; // prevent out of bounds
    // }

    __shared__ int block_changed;
    __shared__ int is_globally_stable;

    while (true) {
        if (threadIdx.x == 0) {
            block_changed = 0;
            is_globally_stable = 0;
        }
        __syncthreads();

        if (m_idx < n && male_match[m_idx] == -1) { // no match
            block_changed = 1;
            *is_stable = 0; // not stable

            int w_idx = men_pref[m_idx * n + propose_next[m_idx]];
        
            bool getLock = false;
            do {
                if(getLock = atomicCAS(&women_lock[w_idx], 0, 1) == 0) {
                    if(woman_match[w_idx] == -1) {
                        woman_match[w_idx] = m_idx;
                        male_match[m_idx] = w_idx;
                    }
                    else if(women_pref[w_idx * n + woman_match[w_idx]] > women_pref[w_idx * n + m_idx]) {
                        male_match[woman_match[w_idx]] = -1;
                        male_match[m_idx] = w_idx;
                        woman_match[w_idx] = m_idx;
                    }
                    propose_next[m_idx]++;
                }
                if(getLock) {
                    atomicExch(&women_lock[w_idx], 0);
                    // atomicCAS(&women_lock[w_idx], 1, 0);
                }
            } while(!getLock);    
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            int flag = (block_changed == 0) ? 1 : 0;
            atomicExch(&d_is_stable_per_block[blockIdx.x], flag);
            __threadfence();           // push it device‐wide

                
                // Wait for all blocks to potentially update their flags
                // This is a naive busy-wait but avoids kernel relaunch
                // for (int wait = 0; wait < 1000; wait++) { } // Small delay
                
                bool done = true;
                for (int i = 0; i < gridDim.x; i++) {
                    int v = atomicAdd(&d_is_stable_per_block[i], 0);
                    if (v == 0) {
                        done = false; 
                        break;
                    }
                }
                if (done) {
                    atomicExch(is_stable_global, 1);
                } else {
                    atomicExch(is_stable_global, 0);
                }
        
            // bool done = true;
            // for (int i = 0; i < gridDim.x; i++) {
            //     int v = atomicAdd(&d_is_stable_per_block[i], 0);
            //     if (v == 0) {
            //         done = false; 
            //         break;
            //     }
            // }
            // if (done) {
            //     atomicExch(is_stable_global, 1);
            // } else {
            //     atomicExch(is_stable_global, 0);
            // }

        }        
        if (threadIdx.x == 0) {
            is_globally_stable = (atomicAdd(is_stable_global,0) != 0);
        }
        __syncthreads();
        if (is_globally_stable) {
            break;
        }
    }
    // __syncthreads();
}

// __global__ void stable_matching_iter(
//     int n,
//     const int *men_pref,    // size n×n
//     const int *women_pref,  // size n×n (rankings)
//     int *male_match,        // size n, init to -1
//     int *woman_match,       // size n, init to -1
//     int *propose_next,      // size n, init to 0
//     int *women_lock,        // size n, init to 0
//     int *d_changed          // single int flag: set to 1 if any change occurs
// ) {    
//     int m_idx = blockIdx.x*blockDim.x + threadIdx.x;
//     if (m_idx >= n) return;

//     if (m_idx < n && male_match[m_idx] == -1) { // no match
//         atomicExch(d_changed, 1);

//         int w_idx = men_pref[m_idx * n + propose_next[m_idx]];
//         bool getLock = false;
//         do {
//             if(getLock = atomicCAS(&women_lock[w_idx], 0, 1) == 0) {
//                 if(woman_match[w_idx] == -1) {
//                     woman_match[w_idx] = m_idx;
//                     male_match[m_idx] = w_idx;
//                 }
//                 else if(women_pref[w_idx * n + woman_match[w_idx]] > women_pref[w_idx * n + m_idx]) {
//                     male_match[woman_match[w_idx]] = -1;
//                     male_match[m_idx] = w_idx;
//                     woman_match[w_idx] = m_idx;
//                 }
//                 propose_next[m_idx]++;
//             }
//             if(getLock) {
//                 atomicExch(&women_lock[w_idx], 0);
//                 // atomicCAS(&women_lock[w_idx], 1, 0);
//             }
//         } while(!getLock);    
//     }
// }

__global__ void stable_matching_iter(
    int n,
    const int *men_pref,    // size n×n
    const int *women_pref,  // size n×n (rankings)
    int *male_match,        // size n, init to -1
    int *woman_match,       // size n, init to -1
    int *propose_next,      // size n, init to 0
    int *women_lock,        // size n, init to 0
    int *d_changed          // single int flag: set to 1 if any change occurs
) {    
    int m_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (m_idx >= n) return;

    if (male_match[m_idx] == -1 && propose_next[m_idx] < n) { // no match and still has women to propose to
        atomicExch(d_changed, 1);

        // Try to propose until either get accepted or run out of women
        bool proposalHandled = false;
        while (!proposalHandled && propose_next[m_idx] < n) {
            int w_idx = men_pref[m_idx * n + propose_next[m_idx]];
            bool getLock = false;
            do {
                if(getLock = atomicCAS(&women_lock[w_idx], 0, 1) == 0) {
                    if(woman_match[w_idx] == -1) {
                        woman_match[w_idx] = m_idx;
                        male_match[m_idx] = w_idx;
                        proposalHandled = true;
                    }
                    else if(women_pref[w_idx * n + woman_match[w_idx]] > women_pref[w_idx * n + m_idx]) {
                        male_match[woman_match[w_idx]] = -1;
                        male_match[m_idx] = w_idx;
                        woman_match[w_idx] = m_idx;
                        proposalHandled = true;
                    }
                    propose_next[m_idx]++;
                }
                if(getLock) {
                    atomicExch(&women_lock[w_idx], 0);
                    // atomicCAS(&women_lock[w_idx], 1, 0);
                }
            } while(!getLock);    
        }
    }
}


__global__ void stable_matching_block(int n, const int* men_pref, const int* women_pref,
                                      int* male_match, int* woman_match, int* propose_next, int* women_lock) {
    
    // for blocked
    int chunk   = (n + blockDim.x - 1) / blockDim.x;
    int start = threadIdx.x * chunk;
    int end   = min(start + chunk, n);

    __shared__ bool block_changed;

    while (true) {
        if (threadIdx.x == 0) {
            block_changed = false;
        }
        __syncthreads();

        // for (int m_idx = start; m_idx < end; ++m_idx) { // block
        for (int m_idx = threadIdx.x; m_idx < n; m_idx+=blockDim.x) { // interleaved
            if (male_match[m_idx] == -1 && propose_next[m_idx] < n) {
                block_changed = true;
                int p = propose_next[m_idx];
                int w_idx = men_pref[m_idx*n + p];

                bool getLock = false;
                do {
                    if(getLock = atomicCAS(&women_lock[w_idx], 0, 1) == 0) {
                        if(woman_match[w_idx] == -1) {
                            woman_match[w_idx] = m_idx;
                            male_match[m_idx] = w_idx;
                        }
                        else if(women_pref[w_idx * n + woman_match[w_idx]] > women_pref[w_idx * n + m_idx]) {
                            male_match[woman_match[w_idx]] = -1;
                            male_match[m_idx] = w_idx;
                            woman_match[w_idx] = m_idx;
                        }
                        propose_next[m_idx]++;
                    }
                    if(getLock) {
                        atomicExch(&women_lock[w_idx], 0);
                        // atomicCAS(&women_lock[w_idx], 1, 0);
                    }
                } while(!getLock);
            }
        }

        __syncthreads();

        if (!block_changed) break;
        __syncthreads();
    }
}



bool is_stable_func(const std::vector<int>& men_data, const std::vector<int>& women_data, const std::vector<int>& men_match, int n) {
    int cnt = 0;

    for (int m = 0; m < n; ++m) {
        int w = men_match[m];
        if (w == -1) {
            printf("Instability: woman %d has no partner\n", m);
            return false;
        }

        for (int i = 0; i < n; ++i) {
            cnt++;
            int preferred_w = men_data[m * n + i];
            if (preferred_w == w) {
                break;
            }

            // find women's partner
            int her_current = -1;
            for (int other_m = 0; other_m < n; ++other_m) {
                if (men_match[other_m] == preferred_w) {
                    her_current = other_m;
                    break;
                }
            }
            if (her_current == -1) {
                printf("Instability: woman %d has no partner\n", her_current);
                return false;
            }

            int m_rank = women_data[preferred_w * n + m];
            int other_rank = women_data[preferred_w * n + her_current];
            if (m_rank < other_rank) {
                printf("Instability: man %d prefers woman %d, and she prefers him over her current partner %d\n",
                        m, preferred_w, her_current);
                return false;
            }
        }
    }

    return true;
}

int main(int argc, char** argv) {
    int opt;
    int n;
    std::string mode;
    while ((opt = getopt(argc, argv, "m:n:")) != -1) {
        switch (opt) {
            case 'n':
                n = atoi(optarg);
                break;
            case 'm':
                mode = optarg;
                break;
            default:
                std::cerr << "Usage: " << argv[0] << " -f input_filename\n";
                exit(EXIT_FAILURE);
        }
    }

    const auto init_start = std::chrono::steady_clock::now();
    
    std::vector<int> men_data(n * n);
    std::vector<int> women_data(n * n);
    // generate random preferences
    for (int i = 0; i < 2 * n; i++) {
        std::vector<int> prefs(n);
        for (int j = 0; j < n; ++j) {
            prefs[j] = j;
        }
        std::mt19937 rng(i * 1000 + 10);
        std::shuffle(prefs.begin(), prefs.end(), rng);

        if (i < n) {
            // man i, prefs[j] is j-th preference
            for (int j = 0; j < n; ++j) {
                men_data[i * n + j] = prefs[j];
            }
        } else {
            // woman i-n -> inverse ranking
            int w = i - n;
            for (int rank = 0; rank < n; ++rank) {
                int j = prefs[rank];
                women_data[w * n + j] = rank;
            }
        }
    }

    const double init_time = std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - init_start).count();
    std::cout << "Initialization time (sec): " << std::fixed << std::setprecision(15) << init_time << '\n';
    const auto compute_start = std::chrono::steady_clock::now();

    int *men_pref, *women_pref, *male_match, *woman_match, *propose_next, *is_stable, *is_stable_global, *women_lock;

    size_t mat_size = n * n;
    hipMalloc(&men_pref, mat_size * sizeof(int));
    hipMalloc(&women_pref, mat_size * sizeof(int));
    hipMalloc(&propose_next, (n) * sizeof(int));
    hipMalloc(&male_match, (n) * sizeof(int));
    hipMalloc(&woman_match, (n) * sizeof(int));
    hipMalloc(&women_lock, (n) * sizeof(int));
    hipMalloc(&is_stable, sizeof(int));
    hipMalloc(&is_stable_global, sizeof(int));

    hipMemcpy(men_pref, men_data.data(), mat_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(women_pref, women_data.data(), mat_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(propose_next, 0, (n) * sizeof(int));
    hipMemset(male_match, -1, (n) * sizeof(int));
    hipMemset(woman_match, -1, (n) * sizeof(int));
    hipMemset(women_lock, 0, (n) * sizeof(int));
    int one = 1;
    hipMemcpy(is_stable, &one, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(is_stable_global, &one, sizeof(int), hipMemcpyHostToDevice);

    int threads_per_block = 256; // TODO
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    int *d_is_stable_per_block;
    hipMalloc(&d_is_stable_per_block, num_blocks * sizeof(int));
    hipMemset(d_is_stable_per_block, 0, (num_blocks) * sizeof(int));
    int *d_global_converged;
    hipMalloc(&d_global_converged, sizeof(int));
    hipMemset(d_global_converged, 0, sizeof(int));
    
    // kernel
    if (mode == "p") {
        stable_matching<<<num_blocks, threads_per_block>>>(n, men_pref, women_pref, male_match, woman_match, propose_next, is_stable, is_stable_global, women_lock, d_is_stable_per_block);
        // stable_matching<<<num_blocks, threads_per_block>>>(n, men_pref, women_pref, male_match, woman_match, propose_next, women_lock, d_is_stable_per_block, d_global_converged);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA post-sync error: " << hipGetErrorString(err) << std::endl;
        }    
    } else if (mode == "b") {
        int threads = min(n, 1024);
        stable_matching_block<<<1, threads>>>(n, men_pref, women_pref, male_match, woman_match, propose_next, women_lock);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA post-sync error: " << hipGetErrorString(err) << std::endl;
        }    

    } else if (mode == "k") {
        int *d_changed;
        hipMalloc(&d_changed, sizeof(int));
                
        int h_changed;
        int max_iterations = n * n; // Safety limit to prevent infinite loops
        int iteration = 0;
        
        do {
            h_changed = 0;
            hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice);
            
            // Launch one iteration
            stable_matching_iter<<<num_blocks, threads_per_block>>>(
                n, men_pref, women_pref, male_match, woman_match, 
                propose_next, women_lock, d_changed);
            hipDeviceSynchronize();
    
            // Check for kernel errors
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "CUDA error in iteration " << iteration << ": " 
                          << hipGetErrorString(err) << std::endl;
                break;
            }
            
            // Fetch back whether anyone changed
            hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
            
            iteration++;
            if (iteration > max_iterations) {
                std::cerr << "Warning: Reached maximum iterations without convergence\n";
                break;
            }
        } while (h_changed != 0);
        
        std::cout << "Algorithm converged after " << iteration << " iterations\n";
        
        // Clean up
        hipFree(d_changed);    
    }
    
   
    const double compute_time = std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - compute_start).count();
    std::cout << "Computation time (sec): " << compute_time << '\n';

    // copy back data
    std::vector<Participant> participants(n * 2);
    std::vector<int> h_men_match(n);
    hipMemcpy(h_men_match.data(), male_match, n * sizeof(int), hipMemcpyDeviceToHost);
    bool stable = is_stable_func(men_data, women_data, h_men_match, n);

    // bool stable = is_stable_matching(participants, n);
    std::cout << "Stable? cuda " << (stable ? "yes" : "no") << std::endl;

    hipFree(men_pref);
    hipFree(women_pref);
    hipFree(propose_next);
    hipFree(male_match);
    hipFree(woman_match);
    hipFree(women_lock);
    hipFree(is_stable);
    hipFree(is_stable_global);

    return 0;
}

