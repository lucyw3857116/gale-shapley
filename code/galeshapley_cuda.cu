#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <algorithm>
#include <string>
#include <unistd.h>
#include <iomanip>

#include "galeshapley.h"

__global__ void stable_matching_block(int n, const int* men_pref, const int* women_pref, int* male_match, int* woman_match, int* propose_next, int* women_lock) {
    int chunk = (n + blockDim.x - 1) / blockDim.x;
    int start = threadIdx.x * chunk;
    int end = min(start + chunk, n);

    __shared__ bool block_changed;

    while (true) {
        if (threadIdx.x == 0) {
            block_changed = false;
        }
        __syncthreads();

        // for (int m_idx = start; m_idx < end; ++m_idx) { // block
        for (int m_idx = threadIdx.x; m_idx < n; m_idx+=blockDim.x) { // interleaved
            if (male_match[m_idx] == -1 && propose_next[m_idx] < n) {
                block_changed = true;
                int p = propose_next[m_idx];
                int w_idx = men_pref[m_idx*n + p];

                bool getLock = false;
                do {
                    if(getLock = atomicCAS(&women_lock[w_idx], 0, 1) == 0) {
                        if(woman_match[w_idx] == -1) {
                            woman_match[w_idx] = m_idx;
                            male_match[m_idx] = w_idx;
                        }
                        else if(women_pref[w_idx * n + woman_match[w_idx]] > women_pref[w_idx * n + m_idx]) {
                            male_match[woman_match[w_idx]] = -1;
                            male_match[m_idx] = w_idx;
                            woman_match[w_idx] = m_idx;
                        }
                        propose_next[m_idx]++;
                    }
                    if(getLock) {
                        atomicExch(&women_lock[w_idx], 0);
                    }
                } while(!getLock);
            }
        }
        __syncthreads();

        if (!block_changed) {
            break;
        }
        __syncthreads();
    }
}

bool is_stable_func(const std::vector<int>& men_data, const std::vector<int>& women_data, const std::vector<int>& men_match, int n) {
    int cnt = 0;

    for (int m = 0; m < n; ++m) {
        int w = men_match[m];
        if (w == -1) {
            printf("Instability: woman %d has no partner\n", m);
            return false;
        }

        for (int i = 0; i < n; ++i) {
            cnt++;
            int preferred_w = men_data[m * n + i];
            if (preferred_w == w) {
                break;
            }

            // find women's partner
            int her_current = -1;
            for (int other_m = 0; other_m < n; ++other_m) {
                if (men_match[other_m] == preferred_w) {
                    her_current = other_m;
                    break;
                }
            }
            if (her_current == -1) {
                printf("Instability: woman %d has no partner\n", her_current);
                return false;
            }

            int m_rank = women_data[preferred_w * n + m];
            int other_rank = women_data[preferred_w * n + her_current];
            if (m_rank < other_rank) {
                printf("Instability: man %d prefers woman %d, and she prefers him over her current partner %d\n",
                        m, preferred_w, her_current);
                return false;
            }
        }
    }

    return true;
}

int main(int argc, char** argv) {
    int opt;
    int n;
    while ((opt = getopt(argc, argv, "m:n:")) != -1) {
        switch (opt) {
            case 'n':
                n = atoi(optarg);
                break;
            default:
                std::cerr << "Usage: " << argv[0] << " -f input_filename\n";
                exit(EXIT_FAILURE);
        }
    }

    const auto init_start = std::chrono::steady_clock::now();
    
    std::vector<int> men_data(n * n);
    std::vector<int> women_data(n * n);
    // generate random preferences
    for (int i = 0; i < 2 * n; i++) {
        std::vector<int> prefs(n);
        for (int j = 0; j < n; ++j) {
            prefs[j] = j;
        }
        std::mt19937 rng(i * 1000 + 10);
        std::shuffle(prefs.begin(), prefs.end(), rng);

        if (i < n) {
            // man i, prefs[j] is j-th preference
            for (int j = 0; j < n; ++j) {
                men_data[i * n + j] = prefs[j];
            }
        } else {
            // woman i-n -> inverse ranking
            int w = i - n;
            for (int rank = 0; rank < n; ++rank) {
                int j = prefs[rank];
                women_data[w * n + j] = rank;
            }
        }
    }

    const double init_time = std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - init_start).count();
    std::cout << "Initialization time (sec): " << std::fixed << std::setprecision(15) << init_time << '\n';
    const auto compute_start = std::chrono::steady_clock::now();

    int *men_pref, *women_pref, *male_match, *woman_match, *propose_next, *women_lock;

    size_t mat_size = n * n;
    hipMalloc(&men_pref, mat_size * sizeof(int));
    hipMalloc(&women_pref, mat_size * sizeof(int));
    hipMalloc(&propose_next, (n) * sizeof(int));
    hipMalloc(&male_match, (n) * sizeof(int));
    hipMalloc(&woman_match, (n) * sizeof(int));
    hipMalloc(&women_lock, (n) * sizeof(int));

    hipMemcpy(men_pref, men_data.data(), mat_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(women_pref, women_data.data(), mat_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(propose_next, 0, (n) * sizeof(int));
    hipMemset(male_match, -1, (n) * sizeof(int));
    hipMemset(woman_match, -1, (n) * sizeof(int));
    hipMemset(women_lock, 0, (n) * sizeof(int));
    
    int threads = min(n, 1024);
    stable_matching_block<<<1, threads>>>(n, men_pref, women_pref, male_match, woman_match, propose_next, women_lock);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA post-sync error: " << hipGetErrorString(err) << std::endl;
    }    

    const double compute_time = std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - compute_start).count();
    std::cout << "Computation time (sec): " << compute_time << '\n';

    // copy back data
    std::vector<Participant> participants(n * 2);
    std::vector<int> h_men_match(n);
    hipMemcpy(h_men_match.data(), male_match, n * sizeof(int), hipMemcpyDeviceToHost);
    bool stable = is_stable_func(men_data, women_data, h_men_match, n);

    // bool stable = is_stable_matching(participants, n);
    std::cout << "Stable? cuda " << (stable ? "yes" : "no") << std::endl;

    hipFree(men_pref);
    hipFree(women_pref);
    hipFree(propose_next);
    hipFree(male_match);
    hipFree(woman_match);
    hipFree(women_lock);

    return 0;
}

